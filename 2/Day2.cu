
#include <hip/hip_runtime.h>
#include <fcntl.h>    /* For O_RDWR */
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#define _GNU_SOURCE

#include <stdio.h>
#include <string.h>
#include <stdlib.h>

struct Position{
	int h;
	int v;
	int aim;
};

/*

__global__ void cuda_hello(struct Position *p){

	printf("hello\n");
    //printf("%d aaaa %d\n",*p.h, *p.v);
	//p->h = 100;
}
*/

int main() {
	
	// nvcc -o a a.cu	

 	FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;

	struct Position p;
	struct Position p2;
	p.h = 0;
	p.v = 0;

	p2.h = 0;
	p2.v = 0;
	p2.aim = 0;
	
    fp = fopen("input.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);

	char dest[1];
    while ((read = getline(&line, &len, fp)) != -1) {
        //printf("Retrieved line of length %zu:\n", read);
        
		// forward down up 
		switch (line[0])
		{
		case 'f':			
			memcpy(dest, line + 7, sizeof(int));	
			p.h = p.h + atoi(dest);
			p2.h = p2.h + atoi(dest);
			p2.v = p2.v + (p2.aim * atoi(dest));
			break;
		case 'u':
			memcpy(dest, line + 2, sizeof(int));
			p.v = p.v - atoi(dest);
			p2.aim = p2.aim - atoi(dest);
			break;
		case 'd':
			memcpy(dest, line + 4, sizeof(int));
			p.v = p.v + atoi(dest);
			p2.aim = p2.aim + atoi(dest);
			break;		
		default:
			break;
		}
    }

    fclose(fp);
    if (line)
        free(line);
	printf("%d",p.h * p.v);
	printf("\n%d",p2.h * p2.v);
    exit(EXIT_SUCCESS);
}