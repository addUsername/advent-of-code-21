
#include <hip/hip_runtime.h>
#include <fcntl.h>    /* For O_RDWR */
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#define _GNU_SOURCE

#include <stdio.h>
#include <string.h>
#include <stdlib.h>

struct Position{
	int h;
	int v;
};

/*

__global__ void cuda_hello(struct Position *p){

	printf("hello\n");
    //printf("%d aaaa %d\n",*p.h, *p.v);
	//p->h = 100;
}
*/

int main() {
	// nvcc -o a a.cu	

 	FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;

	struct Position p;

	p.h = 0;
	p.v = 0;
	
    fp = fopen("input.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);

	char dest[1];
    while ((read = getline(&line, &len, fp)) != -1) {
        //printf("Retrieved line of length %zu:\n", read);
        
		// forward down up 
		switch (line[0])
		{
		case 'f':			
			memcpy(dest, line + 7, sizeof(int));	
			p.h = p.h + atoi(dest);
			break;
		case 'u':
			memcpy(dest, line + 2, sizeof(int));
			p.v = p.v - atoi(dest);
			break;
		case 'd':
			memcpy(dest, line + 4, sizeof(int));
			p.v = p.v + atoi(dest);
			break;		
		default:
			break;
		}
    }

    fclose(fp);
    if (line)
        free(line);
	printf("%d",p.h * p.v);
    exit(EXIT_SUCCESS);
}