
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define DATAXYSIZE 1001

void freeArray(int **a, int m);
int readFileAsStrings(char* filename, char* lines[1000] );
char *strRemove(char *str);
void check(hipError_t err);

__global__ void sumMatrix(int output[][DATAXYSIZE][DATAXYSIZE], int length){

    for(int i = 1; i < length; i++){
        for(int j = 0; j < DATAXYSIZE; j++){
            for(int k = 0; k < DATAXYSIZE; k++){
                output[0][j][k] += output[i][j][k];
            }
        }
    }
    int a = 0;
    for(int j = 0; j < DATAXYSIZE; j++){
        for(int k = 0; k < DATAXYSIZE; k++){
            if (output[0][j][k] > 1){
                a++;
            }
        }
    }
    printf("%d",a);
}
__global__ void drawMoves(int ** boards, int output[][DATAXYSIZE][DATAXYSIZE], bool isEx1 ){

    int idxStart = threadIdx.x * 10;
    int idxFinish = idxStart + 10;

    int x = 0;
    int y = 0;
    int sx = 0;
    int ex = 0;
    int sy = 0;
    int ey = 0;
    
    for(int i = idxStart; i < idxFinish; i++){
        //printf("\n %d %d -> %d %d\n",boards[i][0],boards[i][1],boards[i][2],boards[i][3]);
       
        x = boards[i][2] - boards[i][0];
        y = boards[i][3] - boards[i][1];
        if (isEx1 && x!=0 && y!=0){                
                continue;
        }
        if( x==0 || y==0 ){
            
            sx = boards[i][0];
            ex = boards[i][2];
            if(sx > ex){
                sx = boards[i][2];
                ex = boards[i][0];
            }

            sy = boards[i][1];
            ey = boards[i][3];        
            if(sy>ey){
                sy = boards[i][3];
                ey = boards[i][1];
            }
            
            for(int j = sx; j<ex; j++){
                output[threadIdx.x][ey][j] = output[threadIdx.x][ey][j] + 1;
            }
            
            for(int j = sy; j<=ey; j++){
                output[threadIdx.x][j][ex] =  output[threadIdx.x][j][ex] +1;
            }
        }else{
            sx = boards[i][0];
            ex = boards[i][2];
            
            sy = boards[i][1];
            ey = boards[i][3];
            if(( x==0 || y==0 )){

                if(sx > ex){
                sx = boards[i][2];
                ex = boards[i][0];
                }
                if(sy>ey){
                    sy = boards[i][3];
                    ey = boards[i][1];
                }

                for(int j = sx; j<ex; j++){
                    output[threadIdx.x][ey][j] = output[threadIdx.x][ey][j] + 1;
                }
                
                for(int j = sy; j<=ey; j++){
                    output[threadIdx.x][j][ex] =  output[threadIdx.x][j][ex] +1;
                }
            }else {
                int moveX = 0;
                int moveY = 0;
                int incrX = (x > 0)? 1 : -1;
                int incrY = (y > 0)? 1 : -1;
                for(int j = 0; j<= incrX * x; j++){
                    output[threadIdx.x][sy+moveY][sx+moveX] += 1;
                    moveX += incrX;
                    moveY += incrY;
                }
            }    
        }
    }    
    __syncthreads();
    /*
    for(int i = 0; i<DATAXYSIZE; i++){
        printf("\n");
        for(int j = 0; j<DATAXYSIZE; j++){               
            if(output[threadIdx.x][i][j] == 0){
                printf(".");
            }else{
                printf("%d",output[threadIdx.x][i][j]);
            }               
        }
    }
    */    
   return;
}

int main() {
    
    bool const isAns1 = false;
    int NUMBER_OF_MOVES;
    
    char *lines[1000];
    NUMBER_OF_MOVES = readFileAsStrings("input.txt", lines );    
    printf("%d\n",NUMBER_OF_MOVES);
    int moves[NUMBER_OF_MOVES][4];
    //--------------------Parse Input---------------
    for(int i = 0; i< NUMBER_OF_MOVES; i++){
        strRemove(lines[i]);

        char *ptr = strtok(lines[i], ",");
        if (moves[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        
        moves[i][0] = atoi(ptr);
        moves[i][1] = atoi(strtok(NULL, ","));
        moves[i][2] = atoi(strtok(NULL, ","));
        moves[i][3] = atoi(strtok(NULL, ",")); 
    }
        
    //-----------------Malloc moves in device ---------
    int *ptrDevice[NUMBER_OF_MOVES];
    int **_total;
    for (int i = 0; i< NUMBER_OF_MOVES; i++){

        check( hipMalloc( (void **)&ptrDevice[i], 4 * sizeof(int)));

        check( hipMemcpy(ptrDevice[i], &moves[i], 4 * sizeof(int), hipMemcpyHostToDevice));
        
    }
    check( hipMalloc((void ***)&_total, NUMBER_OF_MOVES*4*sizeof(int)));

    check( hipMemcpy(_total, ptrDevice, NUMBER_OF_MOVES*4*sizeof(int), hipMemcpyHostToDevice));

    //-----------------Malloc array of matrix----------- (good luck here)
    //https://stackoverflow.com/questions/12924155/sending-3d-array-to-cuda-kernel/12925014#12925014
    
    typedef int nRarray[DATAXYSIZE][DATAXYSIZE];
    // overall data set sizes
    const int nz = NUMBER_OF_MOVES/10;
    nRarray *d_c;  // storage for result computed on device
    /*
    // allocate storage for data set
    nRarray *c; // storage for result stored on host
    c = (nRarray *)malloc((nx*ny*nz)*sizeof(int));
    if(c == 0) {
        printf("malloc1 Fail \n");
        return 1;
    }
    */
    // allocate GPU device buffers
    check( hipMalloc((void **) &d_c, (DATAXYSIZE*DATAXYSIZE*nz)*sizeof(int)));

    //-----------------Exec reduce to NUMBER_OF_MOVES / 10-----------------------
    drawMoves<<<1, NUMBER_OF_MOVES/10>>>(_total, d_c, isAns1);
    hipDeviceSynchronize();
    sumMatrix<<<1,1>>>(d_c,nz);
    hipDeviceSynchronize();

    /*
    int *h_out;
    cudaMemcpy(h_out, d_out, sizeof(int *),cudaMemcpyDeviceToHost);
    printf("thiss %d",h_out);
    */
    //-----------------Exec reduce to output matrix-----------------------
    /*
    cudaMemcpy(c, d_c, ((nx*ny*nz)*sizeof(int)), cudaMemcpyDeviceToHost);
    
    cudaCheckErrors("CUDA memcpy failure");
    // and check for accuracy
    for (unsigned i=0; i<nz; i++)
      for (unsigned j=0; j<ny; j++)
        for (unsigned k=0; k<nx; k++)
          if (c[i][j][k] != (i+j+k)) {
            printf("Mismatch at x= %d, y= %d, z= %d  Host= %d, Device = %d\n", i, j, k, (i+j+k), c[i][j][k]);
            return 1;
            }
    printf("Results check!\n");
    */
    hipFree(d_c);    
    return 0;
}

char *strRemove(char *str){
    
    int lenght = strlen(str);

    for (int i = 0; i< lenght; i++){
        if(str[i] == '-'){
            str[i]=',';
        }
        if(str[i] == '>'){
            str[i]=' ';
        }
        str[i] = str[i];
    }
    str[lenght - (1)]='\0';
    return str;
}

void check(hipError_t err){
    if (err != 0) {
        printf("error copying ptrDevice[i] ");
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines[1000]){
    //---------------READING FILE----------------
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    int i = 0;
    // use dos2unix as default, win files text destroy this silently
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
    while (true){
        read = getline(&line, &len, fp);
       
        if (read == -1){
            break;
        }        
        if(strlen(line) < 2){
            continue;
        }
        lines[i] = (char*) malloc(strlen(line));        
        if (lines[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        strcpy(lines[i], line);
        i++;
        
	}
    return i;
}