
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

int readFileAsStrings(char* filename, char* lines[1000] );
char *strRemove(char *str);
void check(hipError_t err);

// this process just one board, so there is a gpu thread per board
__global__ void drawMoves(int ** boards, int SIZE, bool isEx1 ){
    
    
    __shared__ int shared;
    
    //printf("%d",boards[0][1]);
    int idxStart = threadIdx.x * 10;
    int idxFinish = idxStart + 10;
    int d_matrix[10][10];

    
    // ini temp matrix
    for(int i = 0; i<SIZE; i++){
      //  d_matrix[i] = (int*) malloc(SIZE * sizeof(int)); //??
        for(int j = 0; j<SIZE; j++){
            d_matrix[i][j] = 0;
        }
    }
    
    int x = 0;
    int y = 0;
    int sx = 0;
    int ex = 0;
    int sy = 0;
    int ey = 0;
                //0         //10
    for(int i = idxStart; i<idxFinish; i++){
     //   x = boards[i][2] - boards[i][0];
     //   y = boards[i][3] - boards[i][1];
        printf("\n %d %d -> %d %d",boards[i][0],boards[i][1],boards[i][2],boards[i][3]);
        if(isEx1){
            x = boards[i][2] - boards[i][0];
            y = boards[i][3] - boards[i][1];
            if(x!=0 && y!=0){
                printf("skipping");
                continue;
            }
        }
        //Draw horizontal
        sx = boards[i][0]; //8
        ex = boards[i][2]; //0

        if(sx > ex){
            sx = boards[i][2];//0 
            ex = boards[i][0];//8
        }

        for(int j = sx; j<=ex; j++){
            d_matrix[boards[i][3]][j] = d_matrix[boards[i][3]][j] + 1;
        }

        //Draw vertical
        sy = boards[i][1];
        ey = boards[i][3];
        printf("vertical\n");

        if(sy>ey){
            sy = boards[i][3];
            ey = boards[i][1];
        }
        
        for(int j = sy; j<ey; j++){
            d_matrix[j][ex] = d_matrix[j][ex] + 1;
        }

        for(int i = 0; i<SIZE; i++){
            printf("\n");
            for(int j = 0; j<SIZE; j++){
                    printf("%d",d_matrix[i][j]);
            }
        }
        
        printf("\n");

    }

}

int main() {

    
    bool const isAns1 = true;

    int const SIZE = 10;
    int NUMBER_OF_MOVES;
    
    char *lines[1000];
    NUMBER_OF_MOVES = readFileAsStrings("inputa.txt", lines );    
    printf("%s",lines[0]);

    int moves[NUMBER_OF_MOVES][4];
    //--------------------Parse Input---------------
    for(int i = 0; i< NUMBER_OF_MOVES-1; i++){
        strRemove(lines[i]);

        char *ptr = strtok(lines[i], ",");

    
       // moves[i] = (int*) malloc(4*sizeof(int));
        if (moves[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        
        moves[i][0] = atoi(ptr);
        moves[i][1] = atoi(strtok(NULL, ","));
        moves[i][2] = atoi(strtok(NULL, ","));
        moves[i][3] = atoi(strtok(NULL, ",")); 
    }
    printf("%d %d %d %d\n", moves[0][0], moves[0][1], moves[0][2], moves[0][3]);
    

    //-----------------Malloc moves in device ---------
    int *ptrDevice[NUMBER_OF_MOVES];
    int **_total;
    for (int i = 0; i< NUMBER_OF_MOVES-1; i++){

        check( hipMalloc( (void **)&ptrDevice[i], 4 * sizeof(int)));

        check( hipMemcpy(ptrDevice[i], &moves[i], 4 * sizeof(int), hipMemcpyHostToDevice));
    }

    check( hipMalloc((void ***)&_total, NUMBER_OF_MOVES*4*sizeof(int)));

    check( hipMemcpy(_total, ptrDevice, NUMBER_OF_MOVES*4*sizeof(int), hipMemcpyHostToDevice));

    drawMoves<<<1, NUMBER_OF_MOVES/10>>>(_total,SIZE, isAns1);
    hipDeviceSynchronize();

    //-----------------Malloc array of matrix----------- (good luck here)

    //-----------------Exec reduce to NUMBER_OF_MOVES / 10-----------------------

    //-----------------Exec reduce to output matrix-----------------------
    
    
    return 0;




}
char *strRemove(char *str){
    
    int lenght = strlen(str);
   // printf("\n %d \n",lenght);

    for (int i = 0; i< lenght; i++){
        //printf(" %c ",str[i+end]);
        if(str[i] == '-'){
            str[i]=',';
        }
        if(str[i] == '>'){
            str[i]=' ';
        }
        str[i] = str[i];
    }
    str[lenght - (1)]='\0';
    return str;
}

void check(hipError_t err){
    if (err != 0) {
        printf("error copying ptrDevice[i] ");
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines[1000]){
    //---------------READING FILE----------------
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    int i = 0;
    // use dos2unix as default, win files text destroy this silently
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
    // Getting strings from file
    while (true){
        read = getline(&line, &len, fp);
       
        if (read == -1){
            break;
        }        
        // get mem for each string an store its pointer
        if(strlen(line) < 2){
            continue;
        }
        lines[i] = (char*) malloc(strlen(line));        
        if (lines[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        strcpy(lines[i], line);
        i++;
        
	}
    return i;
}