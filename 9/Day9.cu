
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>       // for clock_t, clock(), CLOCKS_PER_SEC

#define X 100 //100
#define Y 100 //100
#define ANS1 false


int readFileAsStrings(char* filename, char* lines[]);
void check(hipError_t err, char *mssg);

__global__ void sum(int* d_out){

    int sum = 0;
    for(int i=0; i<Y; i++){
        sum += d_out[i];
    }
    printf("%d",sum);
}
__global__ void getDeep(int* d_in, int* d_out){

    int idx =  threadIdx.x + blockIdx.x * blockDim.x;
    int col = idx % X;
    int row = idx / X;

    // OKAY this is how you properly print and bool in c
    // printf(isDeeper ? " true " : " false ");
    bool isDeeper = true;

    //left
    if(col > 0){
        isDeeper = ( d_in[idx-1] > d_in[idx])? true: false;
        if(!isDeeper) return;
    }
    //right
    if(col < X-1){
        isDeeper = ( d_in[idx+1] > d_in[idx])? true: false;
        if(!isDeeper) return;
    }
    //up
    if(row > 0){
        isDeeper = ( d_in[idx-X] > d_in[idx])? true: false;
        if(!isDeeper) return;
    }
    //down
    if(row < Y-1){
        isDeeper = ( d_in[idx+X] > d_in[idx])? true: false;
        if(!isDeeper) return;
    }
    atomicAdd(&d_out[row],1+d_in[idx]);
}

int main() {
    //-----------------Read file-----------------------------------------------    
    char *lines[Y];
    int lenLine = readFileAsStrings("input.txt", lines);

    int *board =(int*) calloc(X*Y,sizeof(int*));
    int x = 0;
    int y = 0;
    for(int i=0; i< Y*X; i++){
        y = i / X;
        x = i % X;

        // Casting is like magic
        //char c = lines[y][x];
        //int aux = c - '0';
        //printf("%d",x);

        board[i] = lines[y][x]- '0';
    }
    //-----------------Malloc input--------------------------------------------
    int* d_in;

    check( hipMalloc(&d_in, X*Y*sizeof(int) ),"&d_in");
    check( hipMemcpy(d_in, board, X*Y*sizeof(int),hipMemcpyHostToDevice),"h_in");
    //-----------------Malloc output-------------------------------------------
    int* d_out; // one int for block/row

    check( hipMalloc((int**) &d_out, Y*sizeof(int)), "d_out");

    //---------------Find deep-------------------------------------------------
    clock_t begin = clock();

    getDeep<<<Y,X>>>(d_in, d_out);
    hipDeviceSynchronize();

    sum<<<1,1>>>(d_out);
    hipDeviceSynchronize();

    clock_t end = clock();
    printf("\nThe elapsed time is %f seconds", (double)(end - begin) / CLOCKS_PER_SEC);       
        
    hipFree(d_in);
    hipDeviceReset();

    return 0;
}

void check(hipError_t err, char* mssg){
    if (err != 0) {
        printf("error copying/malloc :%s\n", mssg);
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines[Y]){
    
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
    int i=0;
    while( i<Y ){
        read = getline(&line, &len, fp);
        if (read == -1 ){
            printf("exit");
            exit(EXIT_FAILURE);
        }
        if ( strlen(line) < 2){
            continue;
        }
        lines[i] = (char*) malloc(X*sizeof(char)+1);
        strcpy(lines[i], line);

        //Ugly but..
        strtok(lines[i],"\n");
        i++;
    }
    
    return i;
}