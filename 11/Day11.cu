
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define L 10
#define STEPS 100
#define ANS1 false

int readFileAsStrings(char* filename, char* lines[]);
void check(hipError_t err, char *mssg);

__device__ int sum; // var defined in __device__ persists between kernel calls
__device__ int total_zeros; // part 2

__shared__ bool flag; // this thing resets

__global__ void doStep(int* d_in, int* d_out){

    
    int idx = threadIdx.x;
    int row = idx / L;
    if(idx == 0) flag = true;
    if(idx == 0) total_zeros = 0;
    __syncthreads();    // play with this sync here, it varies the output by a lot

    bool t_flag = false;

    //if (idx != 11) return;
    d_in[idx]++;

    while(flag){

        // i know.. check how others had implemented it
        if(d_in[idx] > 9 && !t_flag){
            //printf("%d ->\n",idx);
            t_flag = true;
            atomicAdd(&sum,1);

            if ( (idx-1)/L == row){
                atomicAdd( &d_in[idx-1],1);
            }        
            if ( (idx+1)/L == row) atomicAdd(&d_in[idx+1],1);
            
            if (idx-L>-1 && (idx-L)/L == row-1 ) atomicAdd(&d_in[idx-L],1);

            if (idx-L+1>-1 && (idx-L+1)/L == row-1 ) atomicAdd(&d_in[idx-L+1],1);

            if (idx-L-1>-1 && (idx-L-1)/L == row-1 ) atomicAdd(&d_in[idx-L-1],1);

            if (idx+L<L*L && (idx+L)/L == row+1) atomicAdd(&d_in[idx+L],1);

            if (idx+L+1<L*L  && (idx+L+1)/L == row+1) atomicAdd(&d_in[idx+L+1],1);

            if (idx+L-1<L*L  && (idx+L-1)/L == row+1) atomicAdd(&d_in[idx+L-1],1);

            
        }
        if(idx == 0) flag = false;
        __syncthreads();
        if(d_in[idx] > 9 && !t_flag){
            flag = true;
        }
        __syncthreads();

        //if(idx == 0) printf("\n 1 more\n");
    }
    if(t_flag) {
        atomicAdd(&total_zeros,1);
        d_in[idx] = 0;
    }
    if(idx == 0){
        d_out[0] = total_zeros;
    }

}

__global__ void show(int* d_in){

    for (int i = 0; i<L; i++){
        for (int j = 0; j<L; j++){

            printf("%d", d_in[i*L+j]);
        }
        printf("\n");
    }
    printf("\n sum: %d\n",sum);
}
 

int main() {
    //-----------------Read file-----------------------------------------------    
    char *lines[L];
    int lenLine = readFileAsStrings("input.txt", lines);

    //---------------itoa()-------------------------------------------------
    // today we go with char, bc we just store 1-9 individually NOPE, atomic ops only work for 32-64 bits, char are not.
    // and it's pretty difficult to implement, you have to do like 2 ops a the same time to not fucked it up
    int *d_in;
    int *h_in = (int*) malloc(L*L*sizeof(int));

    //flattening
    for (int i = 0; i<L; i++){
        for (int j = 0; j<L; j++){
            char string_for_atoi[2] = { lines[i][j], '\0' };
            h_in[i*L+j] = atoi(string_for_atoi);
        }
    }

    check( hipMalloc((int**)&d_in, L*L*sizeof(int)),"&d_in");
    check( hipMemcpy(d_in, h_in, L*L*sizeof(int), hipMemcpyHostToDevice),"d_in");
    free(h_in);

    int* h_out = (int *) malloc(sizeof(int));
    int* d_out;
    check( hipMalloc((int**)&d_out, sizeof(int)),"&d_out");
    //----------------run()-----------------------------------
    clock_t begin = clock();
   
    if(ANS1){
        for(int i =0; i<STEPS; i++){
            doStep<<<1,L*L>>>(d_in, d_out);
            hipDeviceSynchronize();            
        }
        show<<<1,1>>>(d_in);
        hipDeviceSynchronize();
    }else{
            for(int i =0;; i++){
            doStep<<<1,L*L>>>(d_in, d_out);
            hipDeviceSynchronize();
            check( hipMemcpy( h_out, d_out, sizeof(int), hipMemcpyDeviceToHost ),"&d_out");
            if(h_out[0] == 100){
                printf("\nANS2 = %d",i);
                break;
            }
        }
    }
        
    clock_t end = clock();
    printf("\nThe elapsed time is %f seconds", (double)(end - begin) / CLOCKS_PER_SEC);       
        
    hipFree(d_in);
    hipDeviceReset();

    return 0;
}

void check(hipError_t err, char* mssg){
    if (err != 0) {
        printf("error copying/malloc :%s\n", mssg);
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines[L]){
    
    FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
    int i=0;
    while( i<L ){
        read = getline(&line, &len, fp);
        if (read == -1 ){
            printf("exit");
            exit(EXIT_FAILURE);
        }
        if ( strlen(line) < 2){
            continue;
        }
        lines[i] = (char*) malloc(L*sizeof(char)+1);
        strcpy(lines[i], line);

        //Ugly but..
        strtok(lines[i],"\n");
        i++;
    }
    
    return i;
}