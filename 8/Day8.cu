
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>       // for clock_t, clock(), CLOCKS_PER_SEC
#include <math.h>
/* aaaa
  b    c
  b    c  
   dddd 
  e    f
  e    f
   gggg */

#define NUMBER_IN 10 //10
#define LO 4 //# of digits for the output
#define LI 10
#define ANS1 false


int readFileAsStrings(char* filename, char* lines[]);
void check(hipError_t err, char *mssg);


// Get len str
__device__ int len(char* str){

    for (int i =0; i< 10; i++){

        if(str[i] == '\0'){
            return i;
        }
    }
    return NULL;
}
// Returns how many chars from string 1 are present in 2
__device__ int numCoincidences(char* knowStr, char* uknowStr){

    int sum = 0;

    for (int i = 0; i< len(knowStr); i++){
        for (int j =0; j< len(uknowStr); j++){
            
            if (knowStr[i] == uknowStr[j]){
                sum = sum +1;
                break;
            }
        }
    }
    return sum;
}
// Just in this case it's usefull to get the char value from 8 that is not present in 9
__device__ char* specialCaseLen5(char* knowStr, char* uknowStr){


    bool coincidence = false;
    char *toReturn = (char*) malloc(sizeof(char)*2 );
    toReturn[1] = '\0';
    for (int i =0; i< len(knowStr); i++){
        coincidence = false;
        for (int j =0; j< len(uknowStr); j++){
            if (knowStr[i] == uknowStr[j]){
                
                coincidence = true;
            }
        }
        if (!coincidence){
            toReturn[0] = knowStr[i];
            return toReturn;
        }
    }
    return NULL;
}
// Compare 2 strings, order doesn't matter ab == ba
__device__ bool strcmp(char *s1, char *s2){
    
    if(len(s1) != len(s2)){
        return false;
    }

    if(numCoincidences(s1,s2) == len(s1)){
        return true;
    }
    return false;
}
// This func orders array input, position indicates number value
__global__ void getSum(char* d_in){

    int idx = threadIdx.x;
    char* aux = (char*) malloc(sizeof(char)*10*NUMBER_IN);
    
    for(int i = 0; i<(LI); i++){

        //1
        if (len ( d_in+((LO+LI)*idx + i)*10 ) == 2){
            memcpy( aux+(1*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
        }
        //7
        if (len ( d_in+((LO+LI)*idx + i)*10 ) == 3){
            memcpy( aux+(7*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
        }
        //4
        if (len ( d_in+((LO+LI)*idx + i)*10 ) == 4){
            memcpy( aux+(4*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
        }
        //8
        if (len ( d_in+((LO+LI)*idx + i)*10 ) == 7){
            memcpy( aux+(8*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
        }        
    }
    for(int i = 0; i<(LI); i++){

        //0, 9, 6
        if (len ( d_in+((LO+LI)*idx + i)*10 ) == 6){
            
            if ( numCoincidences(aux+(7*10),d_in+((LO+LI)*idx + i)*10) == 2){
                // 6
                memcpy( aux+(6*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);

            }else if ( numCoincidences(aux+(4*10),d_in+((LO+LI)*idx + i)*10) == 4){
                // 9
                memcpy( aux+(9*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
            }else{
                // 0
                memcpy( aux+(0*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
            }
        }
        
    }

    for(int i = 0; i<(LI); i++){

         // 2, 3, 5
        if (len ( d_in+((LO+LI)*idx + i)*10 ) == 5){
            if ( numCoincidences(aux+(1*10),d_in+((LO+LI)*idx + i)*10) == 2){
                // 3
                memcpy( aux+(3*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
            }else {
                char* charWhoHas2butNot9 = specialCaseLen5(aux+(8*10), aux+(9*10));

                if ( numCoincidences(charWhoHas2butNot9,d_in+((LO+LI)*idx + i)*10) == 0){
                    // 5
                    memcpy( aux+(5*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
                }else{
                    // 2
                    memcpy( aux+(2*10), d_in+((LO+LI)*idx + i)*10,sizeof(char)*10);
                }
            }
        }
    }

    for(int i = 0; i<(LI); i++){
        memcpy(  d_in+((LO+LI)*idx + i)*10, aux+(i*10),sizeof(char)*10);
    }
    return;
}

// by using the ordered input array get the output
__global__ void transcribe(char* d_in, int* d_out){

    int idx = threadIdx.x;
    for(int i = LI; i<(LO+LI); i++){
        for(int j = 0; j<(LI); j++){

            // gl, index are hard in a flattened array
            if ( strcmp(d_in+(((LO+LI)*idx + j)*10), d_in+(((LO+LI)*idx + i)*10))){
                atomicAdd(&d_out[j],1);
                break;

            }
        }
    }
}
// For ans2
__global__ void transcribe2(char* d_in, int* d_out){

   
    int idx = threadIdx.x;    
    int number = 10;
    int exp = 3;

    for(int i = LI; i<(LO+LI); i++){
        for(int j = 0; j<(LI); j++){

            // gl, index are hard in a flattened array
            if ( strcmp(d_in+(((LO+LI)*idx + j)*10), d_in+(((LO+LI)*idx + i)*10))){
                number +=(int) pow(10,exp) * j ;
                exp--;
                break;
            }
        }
    }
    printf("%d ",number);
    atomicAdd(&d_out[0],number);
    __syncthreads();
    printf("\nResult= %d ",d_out[0]);
}


int main() {
    //-----------------Read file-----------------------------------------------    
    char *lines[NUMBER_IN];
    
    int lenLine = readFileAsStrings("inputa.txt", lines);
    //-----------------Parse text----------------------------------------------
    // Next time this will be flat (1D) from the beginning   
    char *h_lines[NUMBER_IN][LO+LI];
    char* aux = (char*) malloc(10*sizeof(char));

    for (int i = 0; i<NUMBER_IN; i++ ){ 

        strcpy(aux, strtok(lines[i], " "));
        
        for (int j=0; j<(LO+LI); j++){ 
            h_lines[i][j] = (char*) malloc(10*sizeof(char));

            if(strlen(aux) > 1){                
                strcpy(h_lines[i][j], aux);
            }else{
                strcpy(aux, strtok(NULL, " "));
                strcpy(h_lines[i][j], aux);

            }
            if(j != LO+LI-1 ){
                strcpy(aux, strtok(NULL, " "));
            }
        }
    }

    //-----------------Malloc input--------------------------------------------
    // flattening, kinda cool trick but index become a little bit ugly
    char* d_in;
    char* h_in;
    h_in = (char *) malloc(NUMBER_IN*(LO+LI)*10*sizeof(char));

    for(int i = 0; i<NUMBER_IN; i++){
        for(int j= 0; j< (LO+LI); j++){
            memcpy(h_in+(10*(i*(LO+LI) + j)), h_lines[i][j], 10*sizeof(char));
         }            
    }
    check( hipMalloc(&d_in,NUMBER_IN*(LO+LI)*10*sizeof(char) ),"&d_in");
    check( hipMemcpy(d_in, h_in,NUMBER_IN*(LO+LI)*10*sizeof(char),hipMemcpyHostToDevice),"h_in");

    free(h_in);

    //---------------Order asc input array---------------------------------------
    clock_t begin = clock();
    getSum<<<1,NUMBER_IN>>>(d_in);
    hipDeviceSynchronize();

    int* h_out = (int*) calloc(10,sizeof(int));
    int* d_out;
    check( hipMalloc(&d_out,10*sizeof(int) ),"d_out");
    check( hipMemcpy(d_out, h_out,10*sizeof(int),hipMemcpyHostToDevice),"h_out");

    if(ANS1){
        // This can be improved, just one thread for value..
        transcribe<<<1,NUMBER_IN>>>(d_in, d_out);
        hipDeviceSynchronize();
        
        check( hipMemcpy(h_out, d_out,10*sizeof(int),hipMemcpyDeviceToHost ),"h_out" );

       
        printf("\nAns1 -> %d",h_out[1]+h_out[8]+h_out[7]+h_out[4]);

        free(h_out);
        hipFree(d_out);
    }else{

        transcribe2<<<1,NUMBER_IN>>>(d_in, d_out);
        hipDeviceSynchronize();
        
        check( hipMemcpy(h_out, d_out,10*sizeof(int),hipMemcpyDeviceToHost ),"h_out" );

        printf("%d",d_out[1]);
    }
    clock_t end = clock();
    printf("\nThe elapsed time is %f seconds", (double)(end - begin) / CLOCKS_PER_SEC);
       
        
    hipFree(d_in);
    hipDeviceReset();
    return 0;
}

void check(hipError_t err, char* mssg){
    if (err != 0) {
        printf("error copying/malloc :%s\n", mssg);
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines[NUMBER_IN]){
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
    int i=0;
    while( i<NUMBER_IN ){

        read = getline(&line, &len, fp);
        if (read == -1 || strlen(line) < 2){
            exit(EXIT_FAILURE);
        }
        lines[i] = (char*) malloc(len*sizeof(char));
        strcpy(lines[i], line);

        //Ugly but..
        strtok(lines[i],"\n");
        i++;
    }
        
    return i;
}