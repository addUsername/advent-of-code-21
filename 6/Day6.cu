
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define NUMBER_OF_DAYS 256
#define NUMBER_OF_GROUPS 9
#define TWO 2

int readFileAsStrings(char* filename, char* lines);
void check(hipError_t err, char *mssg);

__global__ void addDays(long** output){

    int nextDayChild = 0;

    for (int i = 0; i<NUMBER_OF_DAYS; i++){
        for (int day = 0; day<9; day++){
            if ( output[day][0] == 0){
                output[day][0] = 8;
                nextDayChild =(day + 7 > 8 )? day - 2 : day + 7 ;
                output[nextDayChild][1] += output[day][1];
            }else{
                output[day][0] -= 1;
            }
        }
    }
    long sum = 0;
    for(int i = 0; i<9; i++){
        sum += output[i][1];
    }
    printf("\nout: %ld ",sum);
}
__global__ void drawMoves(char * input, int length, long **output){

    int day = threadIdx.x;
    int ascciValue = 48 + threadIdx.x;

    for (int i = 0; i <length; i++){
        if ((int) input[i] == ascciValue){
            output[threadIdx.x][1] += 1;
        } 
    }
    __syncthreads();
    
   return;
}

int main() {
    char *lines;
    int const NUMBER_OF_FISH = 600;

    //-----------------Read file-----------------------------------------------
    lines = (char*) malloc(NUMBER_OF_FISH); // NUMBER OF FISH
    int lenLine = readFileAsStrings("input.txt", lines);
    
    //-----------------Malloc input---------------------------------------------
    char* d_lines;
    check( hipMalloc((char**)&d_lines, lenLine * sizeof(char)), "&d_lines");
    check( hipMemcpy(d_lines, lines, lenLine * sizeof(char), hipMemcpyHostToDevice ), "d_lines");

    free(lines);
    //-----------------Malloc output-------------------------------------------
    // array[9][2] -> 9 = total of groups order by its current day before creating another fish
    //                2 = first -> current day / second -> num of fishes
    typedef long nRarray[NUMBER_OF_GROUPS][TWO];
    nRarray *d_total;

    long *ptrDevice[NUMBER_OF_GROUPS];
    long **_total;
    for (int i = 0; i< NUMBER_OF_GROUPS; i++){
        long b[2] = {i,0};
        check( hipMalloc( (void **)&ptrDevice[i], TWO * sizeof(long)), "&ptrDevice");
        check( hipMemcpy(ptrDevice[i], b, TWO*sizeof(long), hipMemcpyHostToDevice), "ptrDevice" );       
    }
    check( hipMalloc((void ***)&_total, NUMBER_OF_GROUPS*TWO*sizeof(long)), "&_total");
    check( hipMemcpy(_total, ptrDevice, NUMBER_OF_GROUPS*TWO*sizeof(long), hipMemcpyHostToDevice), "ptrDevice" );
    
    //-----------------Exec -----------------------
    drawMoves<<<1, NUMBER_OF_GROUPS>>>(d_lines, lenLine, _total);
    hipDeviceSynchronize();
    hipFree(d_lines);

    addDays<<<1,1>>>(_total);
    hipDeviceSynchronize();
    hipFree(_total);

    return 0;
}

void check(hipError_t err, char* mssg){
    if (err != 0) {
        printf("error copying/malloc :%s\n", mssg);
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines){
    //---------------READING FILE----------------
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    int i = 0;
    // use dos2unix as default, win files text destroy this silently
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
        read = getline(&line, &len, fp);
       
        if (read == -1 || strlen(line) < 2){
            exit(EXIT_FAILURE);
        }
        strcpy(lines, line);
        
    return strlen(line);
}