
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define NUMBER_OF_DAYS 9
#define TWO 2

int readFileAsStrings(char* filename, char* lines);
void check(hipError_t err);

__global__ void drawMoves(char * input, int length, int** output){

    int a = (int) input[0];

    int const day = threadIdx.x;
    int const ascciValue = 48 + threadIdx.x;
    output[day][0] = day;
    output[day][1] = 0;

    for (int i = 0; i <length; i++){
        if ((int) input[i] == ascciValue){
            output[threadIdx.x][0] += 1;
        } 
    }

    printf("finish");
    __syncthreads();
    
   return;
}

int main() {
    char *lines;
    int const NUMBER_OF_FISH = 600;

    //-----------------Read file-----------------------------------------------
    lines = (char*) malloc(NUMBER_OF_FISH); // NUMBER OF FISH
    int lenLine = readFileAsStrings("input.txt", lines);
    
    //-----------------Malloc input---------------------------------------------
    char* d_lines;
    check( hipMalloc((char**)&d_lines, lenLine * sizeof(char)));
    check( hipMemcpy(d_lines, lines, lenLine * sizeof(char), hipMemcpyHostToDevice ));

    free(lines);
    //-----------------Malloc output-------------------------------------------
    // array[9][2] -> 9 = total of groups order by its current day before creating another fish
    //                2 = first -> current day / second -> num of fishes
    int h_fish[NUMBER_OF_DAYS][TWO];

    int *ptrDevice[NUMBER_OF_DAYS];
    int **_total;
    for (int i = 0; i< NUMBER_OF_DAYS; i++){

        check( hipMalloc( (void **)&ptrDevice[i], TWO * sizeof(int)));        
    }
    check( hipMalloc((void ***)&_total, NUMBER_OF_DAYS*TWO*sizeof(int)));
    check( hipMemcpy(_total, ptrDevice, NUMBER_OF_DAYS*TWO*sizeof(int), hipMemcpyHostToDevice));
    
    //-----------------Exec -----------------------
    drawMoves<<<1, NUMBER_OF_DAYS>>>(d_lines, lenLine, _total);
    hipDeviceSynchronize();

    return 0;
}

void check(hipError_t err){
    if (err != 0) {
        printf("error copying/malloc ");
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines){
    //---------------READING FILE----------------
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    int i = 0;
    // use dos2unix as default, win files text destroy this silently
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
        read = getline(&line, &len, fp);
       
        if (read == -1 || strlen(line) < 2){
            exit(EXIT_FAILURE);
        }

        //lines = (char*) malloc(strlen(line));        
        if (lines  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        strcpy(lines, line);
        
    return strlen(line);
}