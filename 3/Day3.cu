#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void cuda_hello(char ** b, char* deviceOutput ){

    int NUMBER_ROWS = 1001;
    int sum = 0;

    for (int i = 0; i < NUMBER_ROWS - 1; i++){
        
        if( b[i][threadIdx.x] == '1'){
            sum++;
        }        
    }
    if (sum >= NUMBER_ROWS/2){
        deviceOutput[threadIdx.x] = '1';
    }else{        
        deviceOutput[threadIdx.x] = '0';
    }
    __syncthreads();        
}

//This makes no sense bc just one thread is needed but for practice purposes it's ok
__global__ void getOxigen(char ** b, char* deviceOutput, char* oxi){

    const int NUMBER_ROWS = 1001;
    int LENGHT_ROW = 12+1;

    char *sorted[NUMBER_ROWS];
    int size = NUMBER_ROWS;
    int index = 0;


    for (int i = 0; i < LENGHT_ROW ; i++){
        for (int j = 0; j < size- 1; j++){

            if(i == 0){                
                if(b[j][0] == deviceOutput[0] ){
                    sorted[index] = (char*) malloc(LENGHT_ROW);                
                    sorted[index] = b[j];
                    index++;
                }
            }else{
                if(sorted[j][i] == deviceOutput[i] ){              
                    sorted[index] = sorted[j];
                    index++;
                    
                }
            }
        }
        size = index;
        index = 0;       
    }
    //strcpy doesn't work here
    for(int k = 0; k<LENGHT_ROW; k++){
        oxi[k] = sorted[0][k];
    }    
    __syncthreads();        
}

__global__ void getC02(char ** b, char* deviceOutput, char* co2){

    const int NUMBER_ROWS = 1001;
    int LENGHT_ROW = 12+1;

    char *sorted[NUMBER_ROWS];
    int size = NUMBER_ROWS;
    int index = 0;

    for (int i = 0; i < LENGHT_ROW ; i++){
        for (int j = 0; j < size- 1; j++){

            if(i == 0){                
                if(b[j][0] != deviceOutput[0] ){
                    sorted[index] = (char*) malloc(LENGHT_ROW);                
                    sorted[index] = b[j];
                    index++;
                }
            }else{
                if(sorted[j][i] == deviceOutput[i] ){                
                    sorted[index] = sorted[j];
                    index++;
                    
                }
            }
        }       
        size = index;
        index = 0;       
    }
    //strcpy doesn't work here    
    for(int k = 0; k<LENGHT_ROW; k++){
        co2[k] = sorted[0][k];
    }
    __syncthreads();        
}

int main() {

    int NUMBER_ROWS = 1001;
    int LENGHT_ROW = 12+1;


 	FILE * fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;

    fp = fopen("input.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);	

    
	// this is an array of pointers or String[]
    // if you want to acces to its values cast (char **) b[i]
	// https://stackoverflow.com/a/8824682/13771772
    char *b[NUMBER_ROWS];

    int i = 0;
    // Getting strings from file
    while ((read = getline(&line, &len, fp)) != -1) {

        // get mem for each string an store its pointer
        b[i] = (char*) malloc(LENGHT_ROW);        
        if (b[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }

        //get rid off /n by finishing the line wit thst char.. but i think we are storing the \n and null char, and that's not good
        line[LENGHT_ROW-2] = '\0';
        strcpy(b[i], line);
        i++;
	}
    
    fclose(fp);
    if (line){
		free(line);
	}

    // https://forums.developer.nvidia.com/t/is-copying-an-array-of-character-strings-to-device-memory-absolutely-impossible/17273/11
    // What we want to do is to make our *b[] (aka String[]) visible on device(gpu)
    // This shit is not trivial.
    // cudaMempcy or memcpy doesn't do any kind of "deep copying" so, we need to iterate (over *b[]) to allocate and
    // copy each string, while storing its device adresses in a host array (* ptrDevice[]), then copy this array from host memory into device memory.

    // This holds ptr addresses from device (gpu)    
    char *ptrDevice[NUMBER_ROWS];
    // This is the argument for the cuda func
    char **_total;

    for (int i = 0; i< NUMBER_ROWS -1; i++){

        // malloc 13 chars size and store its "device" pointer to our host ptrDevice
        hipError_t err = hipMalloc((void **)&ptrDevice[i], LENGHT_ROW);
        // _________________________________^_^
        if (err != 0) {
            printf("error allocating");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }
    
        // Then we copy strings from *b[] in the gpu address allocated before
        err = hipMemcpy(ptrDevice[i], b[i], LENGHT_ROW, hipMemcpyHostToDevice);
        if (err != 0) {
            printf("error copying ptrDevice[i] ");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }
        
    }
    // Once done, we need to allocate space for the array with 1000 gpu address.
    hipMalloc((void ***)&_total, LENGHT_ROW*NUMBER_ROWS*sizeof(char));
    // ______________^___^

    // And copy to device
    hipError_t err = hipMemcpy(_total, ptrDevice, LENGHT_ROW*NUMBER_ROWS*sizeof(char), hipMemcpyHostToDevice);
    if (err != 0) {
            printf("error copying _total");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }
        
    char* deviceOutput;
    
    err = hipMalloc((char**) &deviceOutput,LENGHT_ROW * sizeof(char));
    if (err != 0) {
            printf("error allocating");
            printf("%s",hipGetErrorString(err));
            return -1;            
    }

    // Ok Now we are ready to call cuda, we want 1 block of 13 threads bc yes
	// threads < blocks < grid
    // https://gist.github.com/dpiponi/1502434
    // https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/
	cuda_hello<<<1,LENGHT_ROW-1>>>(_total, deviceOutput);

    // This waits for gpu threads to finish
    hipDeviceSynchronize();

    // Here we need to get the output..
    char* gamma = (char*) malloc(LENGHT_ROW);

    //an illegal memory access was encountered
    err = hipMemcpy(gamma, deviceOutput, LENGHT_ROW, hipMemcpyDeviceToHost);
    if (err != 0) {
            printf("error copying gamma ");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }

    char* epsilon = (char*) malloc(LENGHT_ROW);
    for (int i =0; i< LENGHT_ROW -1; i++){
        if(gamma[i] == '1'){
            epsilon[i] = '0';
        }else{
            epsilon[i] = '1';
        }
    }

    printf("\n%s", gamma);                  //Should be 001100001011
    printf("\n%d", strtol(gamma,NULL,2));   //Should be 779 
    printf("\n%s", epsilon);                //Should be 110011110100
    printf("\n%d", strtol(epsilon,NULL,2)); //Should be 3316

    // I really dont know why i have to do gamma+1 and epsilon-1, some binary shit i hope
    printf("\n %ld", (1 + strtol(gamma,NULL,2)) * ( -1 + strtol(epsilon,NULL,2))); //Should be 2583164


    printf("\nSECOND EX");
    char * oxigenDevice;
    err = hipMalloc((void **)&oxigenDevice, LENGHT_ROW);
    // _____________________^_^
    if (err != 0) {
        printf("error allocating");
        printf("%s",hipGetErrorString(err));
        return -1;            
    }

    getOxigen<<<1,1>>>(_total, deviceOutput, oxigenDevice);
    hipDeviceSynchronize();
    
    char * oxigen = (char*) malloc(LENGHT_ROW);
    err = hipMemcpy(oxigen, oxigenDevice, LENGHT_ROW, hipMemcpyDeviceToHost);
    if (err != 0) {
            printf("error copying oxi ");
            printf("%s",hipGetErrorString(err));
            return -1;            
    }
    printf("\n oxigen -> ");
    printf("%s",oxigen);
    printf("\n%ld",strtol(oxigen,NULL,2));
    
    char * co2Device;
    err = hipMalloc((void **)&co2Device, LENGHT_ROW);
    // _____________________^_^
    if (err != 0) {
        printf("error allocating");
        printf("%s",hipGetErrorString(err));
        return -1;
    }
    getC02<<<1,1>>>(_total, deviceOutput, co2Device);
    hipDeviceSynchronize();

    char * co2 = (char*) malloc(LENGHT_ROW);
    err = hipMemcpy(co2, co2Device, LENGHT_ROW, hipMemcpyDeviceToHost);
    if (err != 0) {
            printf("error copying co2 ");
            printf("%s",hipGetErrorString(err));
            return -1;            
    }
    printf("\n c02 -> ");
    printf("%s",co2);    
    printf("\n%ld",strtol(co2,NULL,2) );

    printf("\n %ld",(1+strtol(oxigen,NULL,2)) * (-1+strtol(co2,NULL,2)));
    // check for null before..
    /*
    cudaFree(ptrDevice);
    cudaFree(_total);
    cudaFree(deviceOutput);
    */
    hipDeviceReset();
    hipDeviceSynchronize();    
    
    return 0;	
}
