#include "hip/hip_runtime.h"
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void bingo(int * numbers[], int ** boards, int * finish, int numbersMaxLength, int LINE_LENGTH){

    __shared__ int shared;
    // printf("\n thread %d\n", threadIdx.x);
    int idxStart = (threadIdx.x * 5);
    int idxFinish = idxStart + 5;
    int numOfAppears = 0;
    int round = 5;
    int * line[5];
    int count = 0;

    if(0 != threadIdx.x ){
        return;
    }
     // Yep, we repeat 1º for bucle adding 1 iteration till round == numbersMaxLength
    while(round < numbersMaxLength){

        for(int j=round; j<=round ;j++ ){
            for(int i = idxStart; i < idxFinish; i++){
                //line = boards[i];
                for (int k = 0; k <LINE_LENGTH; k++){
                    
                    printf("  %d ", &boards[i][j]);
                    //printf(" %d",*numbers[k]);
                    
                    // Here i should check if number exists in row
                }
                printf("\n");
                //
            }
            printf("\nNumber -> :");
            
            return;
        }
        round++;
    }
    
    

    
    if(threadIdx.x == 1){
        shared = threadIdx.x;
        *finish = threadIdx.x;
        return;
    }
    __syncthreads();
    if(shared != NULL){
        //printf("return from thread: %d", threadIdx.x);
        return;
    }
}

int main() {

    int const NUMBER_OF_BOARDS = 3;
    int const LENGTH_ROW = 15;
    int NUMBERS_COUNT = 0; //??
    int NUMBER_ROWS = 0;   //??
    hipError_t err;

    //---------------READING FILE----------------
 	FILE * fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;

    fp = fopen("input.txt", "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);	
    
    char *b[1000];
    int i = 0;
    // Getting strings from file
    while ((read = getline(&line, &len, fp)) != -1) {

        // get mem for each string an store its pointer
        if(strlen(line) < 2){
            continue;
        }
        b[i] = (char*) malloc(strlen(line));        
        if (b[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        strcpy(b[i], line);
        i++;
	}
    NUMBER_ROWS = i;
    
    fclose(fp);
    if (line){
		free(line);
	}
    
    //---------------GETTING NUMBERS----------------
    int numbers[100];
	int init_size = strlen(b[0]);
	char *ptr = strtok(b[0], ",");

    int j = 0;
    for(j = 0; true; j++){        
        if(ptr == NULL){
            break;
        }
        numbers[j] = atoi(ptr);
        ptr = strtok(NULL, ",");
    }
    NUMBERS_COUNT = j;
    //---------------MALLOC NUMBERS----------------  
    int *ptrNumbers[NUMBERS_COUNT];
    int **_totalNumbers;
    for (int i = 0; i< NUMBERS_COUNT-1; i++){

        err = hipMalloc((void **)&ptrNumbers[i], sizeof(int));
        if (err != 0) {
            printf("error allocating");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }
                                        // ???
        err = hipMemcpy(ptrNumbers[i], &numbers[i], sizeof(int), hipMemcpyHostToDevice);
        if (err != 0) {
            printf("error copying ptrDevice[i] ");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }        
    }
    
    hipMalloc((void ***)&_totalNumbers, NUMBERS_COUNT*sizeof(int));
    err = hipMemcpy(_totalNumbers, ptrNumbers,  NUMBERS_COUNT*sizeof(int), hipMemcpyHostToDevice);
    if (err != 0) {
            printf("error copying _totalNumbers");
            printf("%s",hipGetErrorString(err));
            return -1;            
    }
    //---------------GETTING BOARD-----------------
    int *boards[NUMBER_ROWS];
    int aux[5];
    for (int j=0; j < NUMBER_ROWS-1; j++){
        ptr = strtok(b[j+1], " ");
        if(ptr == NULL){
                break;
        }
        boards[j] = (int*) malloc(5*sizeof(int));
        for ( i = 0; i<5; i++){

            memcpy(*boards[j][i], atoi(ptr), 5*sizeof(int));
        }
        
        
       //printf("%d", &boards[0][0]);
    }
    
    //---------------MALLOC BOARDS----------------
    int *ptrDevice[NUMBER_ROWS];
    int **_total;
    for (int i = 0; i< NUMBER_ROWS-1; i++){

        err = hipMalloc((void **)&ptrDevice[i], LENGTH_ROW * sizeof(int));
        if (err != 0) {
            printf("error allocating");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }
    
        err = hipMemcpy(ptrDevice[i], &boards[i], LENGTH_ROW * sizeof(int), hipMemcpyHostToDevice);
        if (err != 0) {
            printf("error copying ptrDevice[i] ");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }        
    }
    hipMalloc((void ***)&_total, LENGTH_ROW*NUMBER_ROWS*sizeof(int));
    err = hipMemcpy(_total, ptrDevice, LENGTH_ROW*NUMBER_ROWS*sizeof(char), hipMemcpyHostToDevice);
    if (err != 0) {
            printf("error copying _total");
            printf("%s",hipGetErrorString(err));
            return -1;
    }
    //---------------MALLOC BINGO-----------------
    // if set, its value marks which thread has made bingo
    int* threadBingo;
    err = hipMalloc((void**)&threadBingo, sizeof( int));
    if (err != 0) {
            printf("error allocating threadBingo");
            printf("%s",hipGetErrorString(err));
            return -1;            
    }

    //---------------CALL DEVICE-----------------
    bingo<<<1,NUMBER_OF_BOARDS>>>(_totalNumbers, _total, threadBingo, NUMBERS_COUNT, LENGTH_ROW );
    hipDeviceSynchronize();

    int* threadWhoMadeBingo = (int *) malloc(sizeof(int));
    /*
    err = hipMemcpy(threadWhoMadeBingo, threadBingo, sizeof(int), hipMemcpyDeviceToHost);
    if (err != 0) {
            printf("error copying co2 ");
            printf("%s",hipGetErrorString(err));
            return -1;
    }
    printf("\n thread who made bingo %d", *threadWhoMadeBingo);
    */
    
    hipDeviceReset();
    hipDeviceSynchronize();
    
    return 0;
}
