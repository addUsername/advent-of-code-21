
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>   /* For open(), creat()   */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>


// this process just one board, so there is a gpu thread per board
__global__ void bingo(int * numbers[], int ** boards, int numbersMaxLength, int LINE_LENGTH, bool Ans1){

    
    __shared__ int shared;
    
    int idxStart = threadIdx.x * 5;
    int idxFinish = idxStart + 5;
    int numOfAppearsLine[] = {0,0,0,0,0};
    int numOfAppearsColumn[] = {0,0,0,0,0};
    int round = 0;
    int sum = 0;
    int count = 0;
    
    /*
    if(threadIdx.x > 1 ){
        return;
    }
    */
     // Yep, we repeat 1st for bucle adding 1 iteration till round == numbersMaxLength
     // for num in nums
    while(round < numbersMaxLength){
            // For board line
            for(int i = idxStart; i < idxFinish; i++){
                // for num in line
                for (int k = 0; k <LINE_LENGTH; k++){
                    // Pre calc sum (calculate this for all threads is not cool but)
                    if (round == 0) {
                        sum += boards[i][k];
                    }
                    if( boards[i][k] == *numbers[round]){
                        sum -= boards[i][k];

                       numOfAppearsLine[k] = numOfAppearsLine[k]+1;
                       numOfAppearsColumn[i-idxStart] = numOfAppearsColumn[i-idxStart]+1;
                        if(numOfAppearsLine[k] == 5 ||  numOfAppearsColumn[i-idxStart] == 5 ){
                            printf("bingo!!");
                            printf("\n\n %d",*numbers[round]*sum);
                            shared = *numbers[round]*sum;
                            /*
                            if(Ans1){
                                printf("\nBingoo!");
                                printf("\nnumber: %d",*numbers[round]);
                                printf("\nboard: %d", threadIdx.x);
                                printf("\nline :%d",i-idxStart);


                                printf("\n\n %d",*numbers[round]*sum);
                            }*/
                            return;
                        }                        
                    }
                    __syncthreads();
                    // If Ans 2 just keep going
                    if(Ans1 && shared != NULL){
                        return;
                    }
                }
                __syncthreads();        
            }
        // Add new num and restart all
        round++;
    }
    printf("\nNo Bingo??");
    // last bingoed
    printf("\n %d",shared);
    __syncthreads();
    
}

int main() {

    
    bool const Ans1 = true;
    int const LENGTH_ROW = 5;
    int NUMBERS_COUNT = 0;
    int NUMBER_ROWS = 0;
    int NUMBER_OF_BOARDS = 5 / LENGTH_ROW;
    hipError_t err;

    //---------------READING FILE----------------
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    // use dos2unix as default, win files text destroy this silently
    fp = fopen("input.txt", "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    char *b[1000];
    int i = 0;
    // Getting strings from file
    while (true){
        read = getline(&line, &len, fp);
       
        if (read == -1){
            break;
        }        
        // get mem for each string an store its pointer
        if(strlen(line) < 2){
            continue;
        }
        b[i] = (char*) malloc(strlen(line));        
        if (b[i]  == NULL) {
            printf("unable to allocate memory \n");
            return -1;
        }
        strcpy(b[i], line);
        i++;
        
	}
    printf("%d",i);
    NUMBER_ROWS = i;
    NUMBER_OF_BOARDS = i / 5;
    
    fclose(fp);
    if (line){
		free(line);
	}
    //---------------GETTING NUMBERS----------------
    int numbers[100]; // I see the future
	int init_size = strlen(b[0]);
	char *ptr = strtok(b[0], ",");

    int j = 0;
    for(j = 0; true; j++){        
        if(ptr == NULL){
            break;
        }
        numbers[j] = atoi(ptr);
        ptr = strtok(NULL, ",");
    }
    NUMBERS_COUNT = j;
    //---------------MALLOC NUMBERS----------------  
    int *ptrNumbers[NUMBERS_COUNT];
    int **_totalNumbers;
    for (int i = 0; i< NUMBERS_COUNT-1; i++){

        err = hipMalloc((void **)&ptrNumbers[i], sizeof(int));
        if (err != 0) {
            printf("error allocating");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }

        err = hipMemcpy(ptrNumbers[i], &numbers[i], sizeof(int), hipMemcpyHostToDevice);
        if (err != 0) {
            printf("error copying ptrDevice[i] ");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }        
    }
    
    hipMalloc((void ***)&_totalNumbers, NUMBERS_COUNT*sizeof(int));
    err = hipMemcpy(_totalNumbers, ptrNumbers,  NUMBERS_COUNT*sizeof(int), hipMemcpyHostToDevice);
    if (err != 0) {
            printf("error copying _totalNumbers");
            printf("%s",hipGetErrorString(err));
            return -1;            
    }
    //---------------GETTING BOARD-----------------
    int boards[NUMBER_ROWS][5];
    int aux[5];
    for (int j=0; j < NUMBER_ROWS-1; j++){
        ptr = strtok(b[j+1], " ");
        if(ptr == NULL){
                break;
        }

        for ( i = 0; i<5; i++){

            boards[j][i] = atoi(ptr);
            ptr = strtok(NULL, " ");
        }
    }
    //---------------MALLOC BOARDS----------------
    int *ptrDevice[NUMBER_ROWS];
    int **_total;
    for (int i = 0; i< NUMBER_ROWS-1; i++){

        err = hipMalloc((void **)&ptrDevice[i], LENGTH_ROW * sizeof(int));
        if (err != 0) {
            printf("error allocating");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }
    
        err = hipMemcpy(ptrDevice[i], &boards[i], LENGTH_ROW * sizeof(int), hipMemcpyHostToDevice);
        if (err != 0) {
            printf("error copying ptrDevice[i] ");
            printf("%s",hipGetErrorString(err));
            return -1;            
        }        
    }
    hipMalloc((void ***)&_total, LENGTH_ROW*NUMBER_ROWS*sizeof(int));
    err = hipMemcpy(_total, ptrDevice, LENGTH_ROW*NUMBER_ROWS*sizeof(char), hipMemcpyHostToDevice);
    if (err != 0) {
            printf("error copying _total");
            printf("%s",hipGetErrorString(err));
            return -1;
    }
    bingo<<< 1, NUMBER_OF_BOARDS>>>(_totalNumbers, _total,NUMBERS_COUNT, LENGTH_ROW, Ans1 );
    hipDeviceSynchronize();
 
    hipDeviceReset();
    hipDeviceSynchronize();
    
    return 0;
}
