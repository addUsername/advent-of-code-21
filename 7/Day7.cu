
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>       // for clock_t, clock(), CLOCKS_PER_SEC

#define NUMBER_CRABS 1000
#define WORK_FOR_THREAD 10 //benchmark says this is best
#define ANS1 false

int readFileAsStrings(char* filename, char* lines);
void check(hipError_t err, char *mssg);

__global__ void findLower(int* count, int len){

    int position = -1;
    int min = INT_MAX;
        for(int i=0;i<len-1;i++){
            if(count[i] < min){
                min = count[i];
                position = i;
            }
        }
    printf("\nbest pos=%d, min=%d\n",position, min);    
}

__global__ void getSum(int crabs[], int* count){
    
    int idxStart = WORK_FOR_THREAD * threadIdx.x;
    int idxFinish = idxStart + WORK_FOR_THREAD;
    int position = blockIdx.x;
    int incr = 0;
    
    int a = 0;
    for(int i = idxStart; i < idxFinish; i++){

        a = position - crabs[i];
        a = (a > 0)? a: -1*a;

        if(ANS1){
            incr += a;
        }else{
            incr += (a*(1 + a))/2; // ty Gauss
        }        
    }
    atomicAdd(&count[position],incr);
    __syncthreads();
}

int main() {
    char *lines;
    //-----------------Read file-----------------------------------------------
    lines = (char*) malloc(NUMBER_CRABS*4); // NUMBER OF FISH
    int lenLine = readFileAsStrings("input.txt", lines);
    //-----------------Parse text----------------------------------------------    
    int crabs[NUMBER_CRABS];
    int len = 0;
    int max = 0;
    char* aux = strtok(lines, ",");
    
    while(true){
 
        if (aux  == NULL) {
            break;
        }        
        crabs[len] = atoi(aux);
        max = (max < crabs[len])? crabs[len]: max;
        len++;
        aux = strtok(NULL, ",");
    }
    free(lines);
    //-----------------Malloc input--------------------------------------------
    int* d_lines;
    check( hipMalloc((int**)&d_lines, len * sizeof(int)), "&d_lines");
    check( hipMemcpy(d_lines, crabs, len * sizeof(int), hipMemcpyHostToDevice ), "d_lines");

    //-----------------Atomic operation---------------------------------------
    int *d_count;
    check( hipMalloc( (void **)&d_count, max*sizeof(float)),"d_count" );
    check( hipMemset(d_count, 0, max*sizeof(float)), "count");
    
    clock_t begin = clock();

    getSum<<<max,NUMBER_CRABS/WORK_FOR_THREAD>>>(d_lines, d_count);
    hipDeviceSynchronize();

    clock_t end = clock();
    findLower<<<1,1>>>(d_count,max);

    printf("The elapsed time is %f seconds", (double)(end - begin) / CLOCKS_PER_SEC);

    hipFree(d_lines);
    hipDeviceSynchronize();
    hipFree(d_count);
    return 0;
}

void check(hipError_t err, char* mssg){
    if (err != 0) {
        printf("error copying/malloc :%s\n", mssg);
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines){
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    int i = 0;
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
        read = getline(&line, &len, fp);
       
        if (read == -1 || strlen(line) < 2){
            exit(EXIT_FAILURE);
        }
        strcpy(lines, line);
        
    return strlen(line);
}