
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define NUMBER_CRABS 1000

int readFileAsStrings(char* filename, char* lines);
void check(hipError_t err, char *mssg);

__global__ void findLower(int* count, int len){

    int position = -1;
    int min = INT_MAX;
        for(int i=0;i<len-1;i++){
            if(count[i] < min){
                min = count[i];
                position = i;
            }
        }
    printf("\nbest pos=%d, min=%d\n",position, min);
    
}
__global__ void getSum(int crabs[], int* count){

    //printf("\n t %d | block %d | dim %d",threadIdx.x,blockIdx.x,blockDim.x);
    
    int idxStart = 100 * threadIdx.x;
    int idxFinish = idxStart + 100;
    int position = blockIdx.x;
    int incr = 0;
   // printf("\n%d idxS | idxF %d \n",idxStart,idxFinish);
    
    int a = 0;
    for(int i = idxStart; i < idxFinish; i++){

        a = position - crabs[i];
        incr +=  (a > 0)? a : -1*a;
      //  printf("\n%d a | incr %d \n",a,incr);
    }  
     __syncthreads();
    atomicAdd(&count[position],incr);
    //printf("%d crabs[position] | count %d |  | incr %d\n",crabs[idxStart], count[position],incr);
    __syncthreads();

}

int main() {
    char *lines;
    //-----------------Read file-----------------------------------------------
    lines = (char*) malloc(NUMBER_CRABS*4); // NUMBER OF FISH
    int lenLine = readFileAsStrings("input.txt", lines);
    //-----------------Parse text----------------------------------------------    
    int crabs[NUMBER_CRABS];
    int len = 0;
    int max = 0;
    char* aux = strtok(lines, ",");
    
    while(true){
 
        if (aux  == NULL) {
            break;
        }        
        crabs[len] = atoi(aux);
        max = (max < crabs[len])? crabs[len]: max;
        len++;
        aux = strtok(NULL, ",");
    }
    free(lines);
    //-----------------Malloc input--------------------------------------------
    int* d_lines;
    check( hipMalloc((int**)&d_lines, len * sizeof(int)), "&d_lines");
    check( hipMemcpy(d_lines, crabs, len * sizeof(int), hipMemcpyHostToDevice ), "d_lines");

    //-----------------Atomic operation---------------------------------------
    int *d_count;
    check( hipMalloc( (void **)&d_count, max*sizeof(float)),"d_count" );
    check( hipMemset(d_count, 0, max*sizeof(float)), "count");
        
    getSum<<<max,NUMBER_CRABS/100>>>(d_lines, d_count);
    hipDeviceSynchronize();
    findLower<<<1,1>>>(d_count,max);

    hipFree(d_lines);
    hipDeviceSynchronize();
    hipFree(d_count);
    return 0;
}

void check(hipError_t err, char* mssg){
    if (err != 0) {
        printf("error copying/malloc :%s\n", mssg);
        printf("%s",hipGetErrorString(err));
        exit(err);           
    }
}

int readFileAsStrings(char* filename, char* lines){
 	FILE *fp;
	size_t len = 0;
    char *line = NULL;
    ssize_t read;
    int i = 0;
    fp = fopen(filename, "r");
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
        read = getline(&line, &len, fp);
       
        if (read == -1 || strlen(line) < 2){
            exit(EXIT_FAILURE);
        }
        strcpy(lines, line);
        
    return strlen(line);
}